#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "png_util.h"

#define MAX_N 20000

char plate[2][(MAX_N + 2) * (MAX_N + 2)];
int which = 0;
int n;

__global__ void updatePlate(char *plate, int n, int which) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (i > n || j > n) return;

    int idx = i * (n + 2) + j;
    int base = which * (n + 2) * (n + 2);
    int next = (!which) * (n + 2) * (n + 2);

    int num =
        plate[base + idx - n - 3] +
        plate[base + idx - n - 2] +
        plate[base + idx - n - 1] +
        plate[base + idx - 1] +
        plate[base + idx + 1] +
        plate[base + idx + n + 1] +
        plate[base + idx + n + 2] +
        plate[base + idx + n + 3];

    if (plate[base + idx]) {
        plate[next + idx] = (num == 2 || num == 3) ? 1 : 0;
    } else {
        plate[next + idx] = (num == 3);
    }
}

void print_plate() {
    if (n < 60) {
        for (int i = 1; i <= n; i++) {
            for (int j = 1; j <= n; j++) {
                printf("%d", (int)plate[which][i * (n + 2) + j]);
            }
            printf("\n");
        }
    } else {
        printf("Plate too large to print to screen\n");
    }
    printf("\0");
}

void plate2png(char *filename) {
    unsigned char *img = (unsigned char *)malloc(n * n * sizeof(unsigned char));
    image_size_t sz;
    sz.width = n;
    sz.height = n;

    for (int i = 1; i <= n; i++) {
        for (int j = 1; j <= n; j++) {
            int pindex = i * (n + 2) + j;
            int index = (i - 1) * n + j - 1;
            img[index] = plate[which][pindex] > 0 ? 255 : 0;
        }
    }

    printf("Writing file\n");
    write_png_file(filename, img, sz);
    printf("done writing png\n");
    free(img);
    printf("done freeing memory\n");
}

int main() {
    int M;
    char line[MAX_N];

    if (scanf("%d %d", &n, &M) == 2) {
        size_t platesize = 2 * (n + 2) * (n + 2) * sizeof(char);
        memset(plate[0], 0, platesize / 2);
        memset(plate[1], 0, platesize / 2);

        if (n > 0) {
            for (int i = 1; i <= n; i++) {
                scanf("%s", line);
                for (int j = 0; j < n; j++) {
                    plate[0][i * (n + 2) + j + 1] = line[j] - '0';
                }
            }
        } else {
            n = MAX_N;
            for (int i = 1; i <= n; i++)
                for (int j = 0; j < n; j++)
                    plate[0][i * (n + 2) + j + 1] = rand() % 2;
        }

        // CUDA setup
        char *d_plate;
        hipMalloc(&d_plate, platesize);
        hipMemcpy(d_plate, plate, platesize, hipMemcpyHostToDevice);

        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((n + 15) / 16, (n + 15) / 16);

        for (int i = 0; i < M; i++) {
            printf("\nIteration %d:\n", i);
            updatePlate<<<numBlocks, threadsPerBlock>>>(d_plate, n, which);
            hipDeviceSynchronize();
            which = !which;
        }

        // Copy back to CPU
        hipMemcpy(plate, d_plate, platesize, hipMemcpyDeviceToHost);

        printf("\n\nFinal:\n");
        plate2png("plate.png");
        print_plate();

        // Free device memory
        hipFree(d_plate);
    }

    return 0;
}

